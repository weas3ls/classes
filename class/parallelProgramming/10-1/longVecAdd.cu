
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 2048

__global__ void addvec(int *dc, const int *da, const int *db)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	while (tid < N)
	{
		dc[tid] = da[tid] + db[tid];
		tid += blockDim.x * gridDim.x;
	}
}

int main()
{
	int a[N], b[N], c[N], *da, *db, *dc;
	for (int i = 0; i < N; i++)
	{
		a[i] = -i; b[i] = i*i;
	}
	hipMalloc((void**)& dc, N*sizeof(int));
	hipMalloc((void**)& da, N*sizeof(int));
	hipMalloc((void**)& db, N*sizeof(int));
	hipMemcpy(da, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, b, N*sizeof(int), hipMemcpyHostToDevice);
	addvec<<<(N+127)/128, 128>>>(dc, da, db);
	hipMemcpy(c, dc, N*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dc); hipFree(da); hipFree(db);
	return 0;
}