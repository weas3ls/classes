#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define size 1024

__global__ void sqrtvec(float *b, const float *a)
{
	int i = threadIdx.x;
	b[i] = sqrtf(a[i]);
}

int main()
{
	float a[size];
	for (int i = 0; i < size; i++)
	{
		a[i] = (float)i;
	}
	float b[size] = {0}; float *da, *db;
	hipMalloc((void**)& db, size*sizeof(int));
	hipMalloc((void**)& da, size*sizeof(int));
	hipMemcpy(da, a, size*sizeof(int), hipMemcpyHostToDevice);
	sqrtvec<<<1, size>>>(db, da);
	hipMemcpy(b, db, size*sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < size; i++)
	{
		printf("%f\n", b[i]);
	}
	hipDeviceSynchronize();
	hipFree(db);
	hipFree(da);
	return 0;
}