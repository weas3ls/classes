#include "hip/hip_runtime.h"
#include <stdio.h>

#define size 5

__global__ void addvec(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	const int a[size] = {1, 2, 3, 4, 5};
	const int b[size] = {10, 20, 30, 40, 50};
	int c[size] = {0}; int *da, *db, *dc;
	hipMalloc((void**)& dc, size*sizeof(int));
	hipMalloc((void**)& da, size*sizeof(int));
	hipMalloc((void**)& db, size*sizeof(int));
	hipMemcpy(da, a, size*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, b, size*sizeof(int), hipMemcpyHostToDevice);
	addvec<<<1, size>>>(dc, da, db);
	hipMemcpy(c, dc, size*sizeof(int), hipMemcpyDeviceToHost);
	printf("{1, 2, 3, 4, 5} + {10, 20, 30, 40, 50} = {%d, %d, %d, %d, %d}\n", c[0], c[1], c[2], c[3], c[4]);
	hipDeviceSynchronize();
	hipFree(dc); hipFree(da); hipFree(db);
	return 0;
}