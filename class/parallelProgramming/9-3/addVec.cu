
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

void add(int *a, int *b, int *c)
{
	int tID = 0;
	while (tID < N)
	{
		c[tID] = a[tID] + b[tID];
		tID +=1;
	}
}

int main()
{
	int a[N], b[N], c[N];
	// Fill Arrays
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = 1;
	}
	add(a, b, c);
	for (int i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	return 0;
}